#include "hip/hip_runtime.h"
#include "gasal.h"


enum system_type{
	HOST,
	GPU
};




#define CUDAMALLOCCHECK(error, system) \
		do{\
			err = error;\
			if (hipSuccess != err ) { \
				fprintf(stderr, "Cuda error:%d(%s) at line no. %d in file %s\n", err, hipGetErrorString(err), __LINE__, __FILE__); \
				exit(EXIT_FAILURE);\
			}\
		}while(0)\

#define CUDAMEMCPYCHECK(error, copy_to) \
		do{\
			err = error;\
			if (hipSuccess != err ) { \
				fprintf(stderr, "Cuda error:%d(%s) at line no. %d in file %s\n", err, hipGetErrorString(err), __LINE__, __FILE__); \
				exit(EXIT_FAILURE);\
			}\
		}while(0)\

#define CUDAMEMCPYTOSYMBOLCHECK(error) \
		do{\
			err = error;\
			if (hipSuccess != err ) { \
				fprintf(stderr, "Cuda error:%d(%s) at line no. %d in file %s\n", err, hipGetErrorString(err), __LINE__, __FILE__); \
				exit(EXIT_FAILURE);\
			}\
		}while(0)\


#define CUDAMEMFREECHECK(error) \
		do{\
			err = error;\
			if (hipSuccess != err ) { \
				fprintf(stderr, "Cuda error:%d(%s) at line no. %d in file %s\n", err, hipGetErrorString(err), __LINE__, __FILE__); \
				exit(EXIT_FAILURE);\
			}\
		}while(0)\

#define CUDASTREAMCREATEANDDESTROYCHECK(error) \
		do{\
			err = error;\
			if (hipSuccess != err ) { \
				fprintf(stderr, "Cuda error:%d(%s) at line no. %d in file %s\n", err, hipGetErrorString(err), __LINE__, __FILE__); \
				exit(EXIT_FAILURE);\
			}\
		}while(0)\

#define CUDASTREAMQUERYCHECK(error) \
		do{\
			err = error;\
			if (hipSuccess != err ) { \
				if (err == hipErrorNotReady) return -1; \
				else{\
					fprintf(stderr, "Cuda error:%d(%s) at line no. %d in file %s\n", err, hipGetErrorString(err), __LINE__, __FILE__); \
					exit(EXIT_FAILURE);\
				}\
			}\
		}while(0)\


#define CUDASETDEVICECHECK(error) \
		do{\
			err = error;\
			if (hipSuccess != err ) { \
				fprintf(stderr, "Cuda error:%d(%s) at line no. %d in file %s\n", err, hipGetErrorString(err), __LINE__, __FILE__); \
				exit(EXIT_FAILURE);\
			}\
		}while(0)\

inline int CudaCheckKernelLaunch()
{
	hipError_t err = hipGetLastError();
	if ( hipSuccess != err )
	{
		return -1;

	}

	return 0;
}




#include "gasal_kernels_inl.h"


// The gasal local alignment function without start position computation



void gasal_aln(const uint8_t *batch1, const uint32_t *batch1_lens, const uint32_t *batch1_offsets, const uint8_t *batch2, const uint32_t *batch2_lens, const uint32_t *batch2_offsets,  const uint32_t n_alns, const uint32_t batch1_bytes, const uint32_t batch2_bytes, int32_t *host_aln_score, int32_t *host_batch1_start, int32_t *host_batch2_start, int32_t *host_batch1_end, int32_t *host_batch2_end, int algo, int start) {

	hipError_t err;
	if (n_alns <= 0) {
		fprintf(stderr, "Must perform at least 1 alignment (n_alns > 0)\n");
		exit(EXIT_FAILURE);
	}
	if (batch1_bytes <= 0) {
		fprintf(stderr, "Number of batch1_bytes should be greater than 0\n");
		exit(EXIT_FAILURE);
	}
	if (batch2_bytes <= 0) {
		fprintf(stderr, "Number of batch2_bytes should be greater than 0\n");
		exit(EXIT_FAILURE);
	}

	if (batch1_bytes % 8) {
		fprintf(stderr, "Number of batch1_bytes should be multiple of 8\n");
		exit(EXIT_FAILURE);
	}
	if (batch2_bytes % 8) {
		fprintf(stderr, "Number of batch2_bytes should be multiple of 8\n");
		exit(EXIT_FAILURE);
	}



	uint8_t *unpacked1, *unpacked2;
	uint32_t *packed1_4bit, *packed2_4bit;
	CUDAMALLOCCHECK(hipMalloc(&unpacked1, batch1_bytes * sizeof(uint8_t)), GPU);
	CUDAMALLOCCHECK(hipMalloc(&unpacked2, batch2_bytes * sizeof(uint8_t)), GPU);

	CUDAMALLOCCHECK(hipMalloc(&packed1_4bit, (batch1_bytes/8) * sizeof(uint32_t)), GPU);
	CUDAMALLOCCHECK(hipMalloc(&packed2_4bit, (batch2_bytes/8) * sizeof(uint32_t)), GPU);

	CUDAMEMCPYCHECK(hipMemcpyAsync(unpacked1, batch1, batch1_bytes, hipMemcpyHostToDevice), GPU);
	CUDAMEMCPYCHECK(hipMemcpyAsync(unpacked2, batch2, batch2_bytes, hipMemcpyHostToDevice), GPU);


    uint32_t BLOCKDIM = 128;
    uint32_t N_BLOCKS = (n_alns + BLOCKDIM - 1) / BLOCKDIM;

    int batch1_tasks_per_thread = (int)ceil((double)batch1_bytes/(8*BLOCKDIM*N_BLOCKS));
    int batch2_tasks_per_thread = (int)ceil((double)batch2_bytes/(8*BLOCKDIM*N_BLOCKS));

    gasal_pack_kernel_4bit<<<N_BLOCKS, BLOCKDIM>>>((uint32_t*)(unpacked1),
    						(uint32_t*)(unpacked2), packed1_4bit, packed2_4bit,
    					    batch1_tasks_per_thread, batch2_tasks_per_thread, batch1_bytes/4, batch2_bytes/4);
    hipError_t pack_kernel_err = hipGetLastError();
    if ( hipSuccess != pack_kernel_err )
    {
    	 fprintf(stderr, "Cuda error:%d(%s) at line no. %d in file %s\n", pack_kernel_err, hipGetErrorString(pack_kernel_err), __LINE__, __FILE__);
         exit(EXIT_FAILURE);
    }

    uint32_t *lens1, *lens2, *offsets1, *offsets2;
    CUDAMALLOCCHECK(hipMalloc(&lens1, n_alns * sizeof(uint32_t)), GPU);
    CUDAMALLOCCHECK(hipMalloc(&lens2, n_alns * sizeof(uint32_t)), GPU);
    CUDAMALLOCCHECK(hipMalloc(&offsets1, n_alns * sizeof(uint32_t)), GPU);
    CUDAMALLOCCHECK(hipMalloc(&offsets2, n_alns * sizeof(uint32_t)), GPU);

    CUDAMEMCPYCHECK(hipMemcpyAsync(lens1, batch1_lens, n_alns * sizeof(uint32_t), hipMemcpyHostToDevice), GPU);
    CUDAMEMCPYCHECK(hipMemcpyAsync(lens2, batch2_lens, n_alns * sizeof(uint32_t), hipMemcpyHostToDevice), GPU);
    CUDAMEMCPYCHECK(hipMemcpyAsync(offsets1, batch1_offsets, n_alns * sizeof(uint32_t), hipMemcpyHostToDevice), GPU);
    CUDAMEMCPYCHECK(hipMemcpyAsync(offsets2, batch2_offsets, n_alns * sizeof(uint32_t), hipMemcpyHostToDevice), GPU);




    int32_t *aln_score, *batch1_start, *batch2_start, *batch1_end, *batch2_end;

    CUDAMALLOCCHECK(hipMalloc(&aln_score, n_alns * sizeof(int32_t)), GPU);
    if (algo == GLOBAL) {
    	batch1_start = NULL;
    	batch1_end = NULL;
    	batch2_start = NULL;
    	batch2_end = NULL;
    } else {
    	CUDAMALLOCCHECK(hipMalloc(&batch2_end, n_alns * sizeof(uint32_t)), GPU);
    	if (start == WITH_START) {
    		CUDAMALLOCCHECK(hipMalloc(&batch2_start, n_alns * sizeof(uint32_t)), GPU);
    	} else
    		batch2_start = NULL;
    	if (algo == LOCAL) {
    		CUDAMALLOCCHECK(hipMalloc(&batch1_end, n_alns * sizeof(uint32_t)), GPU);
    		if (start == WITH_START) {
    			CUDAMALLOCCHECK(hipMalloc(&batch1_start, n_alns * sizeof(uint32_t)), GPU);
    		} else
    			batch1_start = NULL;
    	} else {
    		batch1_start = NULL;
    		batch1_end = NULL;
    	}
    }



    if (start == WITH_START) {
		gasal_local_with_start_kernel<<<N_BLOCKS, BLOCKDIM>>>(packed1_4bit, packed2_4bit, lens1,
				lens2, offsets1, offsets2, aln_score,
				batch1_end, batch2_end, batch1_start,
				batch2_start, n_alns);
	} else {
		gasal_local_kernel<<<N_BLOCKS, BLOCKDIM>>>(packed1_4bit, packed2_4bit, lens1,
				lens2, offsets1, offsets2, aln_score,
				batch1_end, batch2_end, n_alns);
	}

    hipError_t aln_kernel_err = hipGetLastError();
    if ( hipSuccess != aln_kernel_err )
    {
    	fprintf(stderr, "Cuda error:%d(%s) at line no. %d in file %s\n", aln_kernel_err, hipGetErrorString(aln_kernel_err), __LINE__, __FILE__);
    	exit(EXIT_FAILURE);
    }

    if (host_aln_score != NULL && aln_score != NULL) CUDAMEMCPYCHECK(hipMemcpy(host_aln_score, aln_score, n_alns * sizeof(int32_t), hipMemcpyDeviceToHost), HOST);
    else {
    		fprintf(stderr, "The *host_aln_score input can't be NULL I am here\n");
    		exit(EXIT_FAILURE);
    }
    if (host_batch1_start != NULL && batch1_start != NULL) CUDAMEMCPYCHECK(hipMemcpy(host_batch1_start, batch1_start, n_alns * sizeof(int32_t), hipMemcpyDeviceToHost), HOST);
    if (host_batch2_start != NULL && batch2_start != NULL) CUDAMEMCPYCHECK(hipMemcpy(host_batch2_start, batch2_start, n_alns * sizeof(int32_t), hipMemcpyDeviceToHost), HOST);
    if (host_batch1_end != NULL && batch1_end != NULL) CUDAMEMCPYCHECK(hipMemcpy(host_batch1_end, batch1_end, n_alns * sizeof(int32_t), hipMemcpyDeviceToHost), HOST);
    if (host_batch2_end != NULL && batch2_end != NULL) CUDAMEMCPYCHECK(hipMemcpy(host_batch2_end, batch2_end, n_alns * sizeof(int32_t), hipMemcpyDeviceToHost), HOST);

    if (unpacked1 != NULL) CUDAMEMFREECHECK(hipFree(unpacked1));
    if (unpacked2 != NULL) CUDAMEMFREECHECK(hipFree(unpacked2));
    if (packed1_4bit != NULL) CUDAMEMFREECHECK(hipFree(packed1_4bit));
    if (packed2_4bit != NULL) CUDAMEMFREECHECK(hipFree(packed2_4bit));
    if (offsets1 != NULL) CUDAMEMFREECHECK(hipFree(offsets1));
    if (offsets2 != NULL) CUDAMEMFREECHECK(hipFree(offsets2));
    if (lens1 != NULL) CUDAMEMFREECHECK(hipFree(lens1));
    if (lens2 != NULL) CUDAMEMFREECHECK(hipFree(lens2));
    if (aln_score != NULL) CUDAMEMFREECHECK(hipFree(aln_score));
    if (batch1_start != NULL) CUDAMEMFREECHECK(hipFree(batch1_start));
    if (batch2_start != NULL) CUDAMEMFREECHECK(hipFree(batch2_start));
    if (batch1_end != NULL) CUDAMEMFREECHECK(hipFree(batch1_end));
    if (batch2_end != NULL) CUDAMEMFREECHECK(hipFree(batch2_end));


    return;
}

gasal_gpu_storage* gasal_aln_async(const uint8_t *batch1, const uint32_t *batch1_lens, const uint32_t *batch1_offsets, const uint8_t *batch2, const uint32_t *batch2_lens, const uint32_t *batch2_offsets,  const uint32_t n_alns, const uint32_t batch1_bytes, const uint32_t batch2_bytes, int algo, int start) {

	hipError_t err;
	if (n_alns <= 0) {
		fprintf(stderr, "Must perform at least 1 alignment (n_alns > 0)\n");
		exit(EXIT_FAILURE);
	}
	if (batch1_bytes <= 0) {
		fprintf(stderr, "Number of batch1_bytes should be greater than 0\n");
		exit(EXIT_FAILURE);
	}
	if (batch2_bytes <= 0) {
		fprintf(stderr, "Number of batch2_bytes should be greater than 0\n");
		exit(EXIT_FAILURE);
	}

	if (batch1_bytes % 8) {
		fprintf(stderr, "Number of batch1_bytes should be multiple of 8\n");
		exit(EXIT_FAILURE);
	}
	if (batch2_bytes % 8) {
		fprintf(stderr, "Number of batch2_bytes should be multiple of 8\n");
		exit(EXIT_FAILURE);
	}


	gasal_gpu_storage *gpu_storage = (gasal_gpu_storage*)calloc(1, sizeof(gasal_gpu_storage));
	hipStream_t str;

	CUDASTREAMCREATEANDDESTROYCHECK(hipStreamCreate(&str));


	CUDAMALLOCCHECK(hipMalloc(&(gpu_storage->unpacked1), batch1_bytes * sizeof(uint8_t)), GPU);
	CUDAMALLOCCHECK(hipMalloc(&(gpu_storage->unpacked2), batch2_bytes * sizeof(uint8_t)), GPU);

	CUDAMALLOCCHECK(hipMalloc(&(gpu_storage->packed1_4bit), (batch1_bytes/8) * sizeof(uint32_t)), GPU);
	CUDAMALLOCCHECK(hipMalloc(&(gpu_storage->packed2_4bit), (batch2_bytes/8) * sizeof(uint32_t)), GPU);

	CUDAMEMCPYCHECK(hipMemcpyAsync(gpu_storage->unpacked1, batch1, batch1_bytes, hipMemcpyHostToDevice, str), GPU);
	CUDAMEMCPYCHECK(hipMemcpyAsync(gpu_storage->unpacked2, batch2, batch2_bytes, hipMemcpyHostToDevice, str), GPU);


    uint32_t BLOCKDIM = 128;
    uint32_t N_BLOCKS = (n_alns + BLOCKDIM - 1) / BLOCKDIM;

    int batch1_tasks_per_thread = (int)ceil((double)batch1_bytes/(8*BLOCKDIM*N_BLOCKS));
    int batch2_tasks_per_thread = (int)ceil((double)batch2_bytes/(8*BLOCKDIM*N_BLOCKS));

    gasal_pack_kernel_4bit<<<N_BLOCKS, BLOCKDIM, 0, str>>>((uint32_t*)(gpu_storage->unpacked1),
    						(uint32_t*)(gpu_storage->unpacked2), gpu_storage->packed1_4bit, gpu_storage->packed2_4bit,
    					    batch1_tasks_per_thread, batch2_tasks_per_thread, batch1_bytes/4, batch2_bytes/4);
    hipError_t pack_kernel_err = hipGetLastError();
    if ( hipSuccess != pack_kernel_err )
    {
    	 fprintf(stderr, "Cuda error:%d(%s) at line no. %d in file %s\n", pack_kernel_err, hipGetErrorString(pack_kernel_err), __LINE__, __FILE__);
         exit(EXIT_FAILURE);
    }


    CUDAMALLOCCHECK(hipMalloc(&(gpu_storage->lens1), n_alns * sizeof(uint32_t)), GPU);
    CUDAMALLOCCHECK(hipMalloc(&(gpu_storage->lens2), n_alns * sizeof(uint32_t)), GPU);
    CUDAMALLOCCHECK(hipMalloc(&(gpu_storage->offsets1), n_alns * sizeof(uint32_t)), GPU);
    CUDAMALLOCCHECK(hipMalloc(&(gpu_storage->offsets2), n_alns * sizeof(uint32_t)), GPU);

    CUDAMEMCPYCHECK(hipMemcpyAsync(gpu_storage->lens1, batch1_lens, n_alns * sizeof(uint32_t), hipMemcpyHostToDevice, str), GPU);
    CUDAMEMCPYCHECK(hipMemcpyAsync(gpu_storage->lens2, batch2_lens, n_alns * sizeof(uint32_t), hipMemcpyHostToDevice, str), GPU);
    CUDAMEMCPYCHECK(hipMemcpyAsync(gpu_storage->offsets1, batch1_offsets, n_alns * sizeof(uint32_t), hipMemcpyHostToDevice, str), GPU);
    CUDAMEMCPYCHECK(hipMemcpyAsync(gpu_storage->offsets2, batch2_offsets, n_alns * sizeof(uint32_t), hipMemcpyHostToDevice, str), GPU);





	CUDAMALLOCCHECK(hipMalloc(&(gpu_storage->aln_score), n_alns * sizeof(int32_t)), GPU);
	if (algo == GLOBAL) {
		gpu_storage->batch1_start = NULL;
		gpu_storage->batch1_end = NULL;
		gpu_storage->batch2_start = NULL;
		gpu_storage->batch2_end = NULL;
	} else {
		CUDAMALLOCCHECK(hipMalloc(&(gpu_storage->batch2_end), n_alns * sizeof(uint32_t)), GPU);
		if (start == WITH_START){
			CUDAMALLOCCHECK(hipMalloc(&(gpu_storage->batch2_start), n_alns * sizeof(uint32_t)), GPU);
		}
		else gpu_storage->batch2_start = NULL;
		if (algo == LOCAL) {
			CUDAMALLOCCHECK(hipMalloc(&(gpu_storage->batch1_end), n_alns * sizeof(uint32_t)), GPU);
			if (start == WITH_START){
				CUDAMALLOCCHECK(hipMalloc(&(gpu_storage->batch1_start), n_alns * sizeof(uint32_t)), GPU);
			}
			else gpu_storage->batch1_start = NULL;
		} else {
			gpu_storage->batch1_start = NULL;
			gpu_storage->batch1_end = NULL;
		}
	}



    if (start == WITH_START) {
		gasal_local_with_start_kernel<<<N_BLOCKS, BLOCKDIM, 0, str>>>(gpu_storage->packed1_4bit, gpu_storage->packed2_4bit, gpu_storage->lens1,
				gpu_storage->lens2, gpu_storage->offsets1, gpu_storage->offsets2, gpu_storage->aln_score,
				gpu_storage->batch1_end, gpu_storage->batch2_end, gpu_storage->batch1_start,
				gpu_storage->batch2_start, n_alns);
	} else {
		gasal_local_kernel<<<N_BLOCKS, BLOCKDIM, 0, str>>>(gpu_storage->packed1_4bit, gpu_storage->packed2_4bit, gpu_storage->lens1,
				gpu_storage->lens2, gpu_storage->offsets1, gpu_storage->offsets2, gpu_storage->aln_score,
				gpu_storage->batch1_end, gpu_storage->batch2_end, n_alns);
	}

    hipError_t aln_kernel_err = hipGetLastError();
    if ( hipSuccess != aln_kernel_err )
    {
    	fprintf(stderr, "Cuda error:%d(%s) at line no. %d in file %s\n", aln_kernel_err, hipGetErrorString(aln_kernel_err), __LINE__, __FILE__);
    	exit(EXIT_FAILURE);
    }

    gpu_storage->str = str;
    return gpu_storage;
}

gasal_error_t gasal_get_aln_async_results(gasal_gpu_storage *gpu_storage, uint32_t n_alns, int32_t *host_aln_score, int32_t *host_batch1_start, int32_t *host_batch2_start, int32_t *host_batch1_end, int32_t *host_batch2_end) {

	hipError_t err;
	CUDASTREAMQUERYCHECK(hipStreamQuery(gpu_storage->str));

	if (host_aln_score != NULL && gpu_storage->aln_score != NULL) CUDAMEMCPYCHECK(hipMemcpy(host_aln_score, gpu_storage->aln_score, n_alns * sizeof(int32_t), hipMemcpyDeviceToHost), HOST);
	else {
		fprintf(stderr, "The *host_aln_score input can't be NULL\n");
		exit(EXIT_FAILURE);
	}
	if (host_batch1_start != NULL && gpu_storage->batch1_start != NULL) CUDAMEMCPYCHECK(hipMemcpy(host_batch1_start, gpu_storage->batch1_start, n_alns * sizeof(int32_t), hipMemcpyDeviceToHost), HOST);
	if (host_batch2_start != NULL && gpu_storage->batch2_start != NULL) CUDAMEMCPYCHECK(hipMemcpy(host_batch2_start, gpu_storage->batch2_start, n_alns * sizeof(int32_t), hipMemcpyDeviceToHost), HOST);
	if (host_batch1_end != NULL && gpu_storage->batch1_end != NULL) CUDAMEMCPYCHECK(hipMemcpy(host_batch1_end, gpu_storage->batch1_end, n_alns * sizeof(int32_t), hipMemcpyDeviceToHost), HOST);
	if (host_batch2_end != NULL && gpu_storage->batch2_end != NULL) CUDAMEMCPYCHECK(hipMemcpy(host_batch2_end, gpu_storage->batch2_end, n_alns * sizeof(int32_t), hipMemcpyDeviceToHost), HOST);

	if (gpu_storage->unpacked1 != NULL) CUDAMEMFREECHECK(hipFree(gpu_storage->unpacked1));
	if (gpu_storage->unpacked2 != NULL) CUDAMEMFREECHECK(hipFree(gpu_storage->unpacked2));
	if (gpu_storage->packed1_4bit != NULL) CUDAMEMFREECHECK(hipFree(gpu_storage->packed1_4bit));
	if (gpu_storage->packed2_4bit != NULL) CUDAMEMFREECHECK(hipFree(gpu_storage->packed2_4bit));
	if (gpu_storage->offsets1 != NULL) CUDAMEMFREECHECK(hipFree(gpu_storage->offsets1));
	if (gpu_storage->offsets2 != NULL) CUDAMEMFREECHECK(hipFree(gpu_storage->offsets2));
	if (gpu_storage->lens1 != NULL) CUDAMEMFREECHECK(hipFree(gpu_storage->lens1));
	if (gpu_storage->lens2 != NULL) CUDAMEMFREECHECK(hipFree(gpu_storage->lens2));
	if (gpu_storage->aln_score != NULL) CUDAMEMFREECHECK(hipFree(gpu_storage->aln_score));
	if (gpu_storage->batch1_start != NULL) CUDAMEMFREECHECK(hipFree(gpu_storage->batch1_start));
	if (gpu_storage->batch2_start != NULL) CUDAMEMFREECHECK(hipFree(gpu_storage->batch2_start));
	if (gpu_storage->batch1_end != NULL) CUDAMEMFREECHECK(hipFree(gpu_storage->batch1_end));
	if (gpu_storage->batch2_end != NULL) CUDAMEMFREECHECK(hipFree(gpu_storage->batch2_end));

	CUDASTREAMCREATEANDDESTROYCHECK(hipStreamDestroy(gpu_storage->str));

	return 0;
}



void gasal_copy_subst_scores(gasal_subst_scores *subst){

	hipError_t err;
	CUDAMEMCPYTOSYMBOLCHECK(hipMemcpyToSymbol(HIP_SYMBOL(_cudaGapO), &(subst->gap_open), sizeof(int32_t), 0, hipMemcpyHostToDevice));
	CUDAMEMCPYTOSYMBOLCHECK(hipMemcpyToSymbol(HIP_SYMBOL(_cudaGapExtend), &(subst->gap_extend), sizeof(int32_t), 0, hipMemcpyHostToDevice));
	int32_t gapoe = subst->gap_open + subst->gap_extend;
	CUDAMEMCPYTOSYMBOLCHECK(hipMemcpyToSymbol(HIP_SYMBOL(_cudaGapOE), &(gapoe), sizeof(int32_t), 0, hipMemcpyHostToDevice));
	CUDAMEMCPYTOSYMBOLCHECK(hipMemcpyToSymbol(HIP_SYMBOL(_cudaMatchScore), &(subst->match), sizeof(int32_t), 0, hipMemcpyHostToDevice));
	CUDAMEMCPYTOSYMBOLCHECK(hipMemcpyToSymbol(HIP_SYMBOL(_cudaMismatchScore), &(subst->mismatch), sizeof(int32_t), 0, hipMemcpyHostToDevice));
	return;
}

void gasal_host_malloc(void *mem_ptr, uint32_t n_bytes) {

	hipError_t err;
	CUDAMALLOCCHECK(hipHostMalloc(&mem_ptr, n_bytes), HOST);
}

void gasal_host_free(void *mem_free_ptr) {

	hipError_t err;
	CUDAMEMFREECHECK(hipHostFree(mem_free_ptr));
}








