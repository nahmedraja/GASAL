#include "hip/hip_runtime.h"
#include "gasal.h"




#define CHECKCUDAERROR(error) \
		do{\
			err = error;\
			if (hipSuccess != err ) { \
				fprintf(stderr, "[GASAL] CUDA error:%d(%s) at line no. %d in file %s\n", err, hipGetErrorString(err), __LINE__, __FILE__); \
				exit(EXIT_FAILURE);\
			}\
		}while(0)\


inline int CudaCheckKernelLaunch()
{
	hipError_t err = hipGetLastError();
	if ( hipSuccess != err )
	{
		return -1;

	}

	return 0;
}




#include "gasal_kernels_inl.h"





//GASAL2 blocking alignment function
void gasal_aln(gasal_gpu_storage_t *gpu_storage, const uint8_t *query_batch, const uint32_t *query_batch_offsets, const uint32_t *query_batch_lens, const uint8_t *target_batch, const uint32_t *target_batch_offsetss, const uint32_t *target_batch_lens, const uint32_t actual_query_batch_bytes, const uint32_t actual_target_batch_bytes, const uint32_t actual_n_alns, int32_t *host_aln_score, int32_t *host_query_batch_start, int32_t *host_target_batch_start, int32_t *host_query_batch_end, int32_t *host_target_batch_end,  int algo, int start) {

	hipError_t err;
	if (actual_n_alns <= 0) {
		fprintf(stderr, "[GASAL] Must perform at least 1 alignment (n_alns > 0)\n");
		exit(EXIT_FAILURE);
	}
	if (actual_query_batch_bytes <= 0) {
		fprintf(stderr, "[GASAL] Number of query_batch_bytes should be greater than 0\n");
		exit(EXIT_FAILURE);
	}
	if (actual_target_batch_bytes <= 0) {
		fprintf(stderr, "[GASAL] Number of target_batch_bytes should be greater than 0\n");
		exit(EXIT_FAILURE);
	}

	if (actual_query_batch_bytes % 8) {
		fprintf(stderr, "[GASAL] Number of query_batch_bytes should be multiple of 8\n");
		exit(EXIT_FAILURE);
	}
	if (actual_target_batch_bytes % 8) {
		fprintf(stderr, "[GASAL] Number of target_batch_bytes should be multiple of 8\n");
		exit(EXIT_FAILURE);

	}
	//--------------if pre-allocated memory is less, allocate more--------------------------
	if (gpu_storage->max_query_batch_bytes < actual_query_batch_bytes) {
		fprintf(stderr, "[GASAL]  max_query_batch_bytes(%d) should be >= acutal_query_batch_bytes(%d)\n", gpu_storage->max_query_batch_bytes, actual_query_batch_bytes);

		int i = 2;
		while ( (gpu_storage->max_query_batch_bytes * i) < actual_query_batch_bytes) i++;

		fprintf(stderr, "[GASAL] Allocating %d bytes on GPU\n", gpu_storage->max_query_batch_bytes*i);
		gpu_storage->max_query_batch_bytes = gpu_storage->max_query_batch_bytes * i;

		if (gpu_storage->unpacked_query_batch != NULL) CHECKCUDAERROR(hipFree(gpu_storage->unpacked_query_batch));
		if (gpu_storage->packed_query_batch != NULL) CHECKCUDAERROR(hipFree(gpu_storage->packed_query_batch));

		CHECKCUDAERROR(hipMalloc(&(gpu_storage->unpacked_query_batch), gpu_storage->max_query_batch_bytes * sizeof(uint8_t)));
		CHECKCUDAERROR(hipMalloc(&(gpu_storage->packed_query_batch), (gpu_storage->max_query_batch_bytes/8) * sizeof(uint32_t)));




	}

	if (gpu_storage->max_target_batch_bytes < actual_target_batch_bytes) {
		fprintf(stderr, "max_target_batch_bytes(%d) should be >= acutal_target_batch_bytes(%d)\n", gpu_storage->max_target_batch_bytes, actual_target_batch_bytes);

		int i = 2;
		while ( (gpu_storage->max_target_batch_bytes * i) < actual_target_batch_bytes) i++;

		fprintf(stderr, "[GASAL] Allocating %d bytes on GPU\n", gpu_storage->max_target_batch_bytes*i);
		gpu_storage->max_target_batch_bytes = gpu_storage->max_target_batch_bytes * i;

		if (gpu_storage->unpacked_target_batch != NULL) CHECKCUDAERROR(hipFree(gpu_storage->unpacked_target_batch));
		if (gpu_storage->packed_target_batch != NULL) CHECKCUDAERROR(hipFree(gpu_storage->packed_target_batch));

		CHECKCUDAERROR(hipMalloc(&(gpu_storage->unpacked_target_batch), gpu_storage->max_target_batch_bytes * sizeof(uint8_t)));
		CHECKCUDAERROR(hipMalloc(&(gpu_storage->packed_target_batch), (gpu_storage->max_target_batch_bytes/8) * sizeof(uint32_t)));


	}

	if (gpu_storage->max_n_alns < actual_n_alns) {
		fprintf(stderr, "[GASAL] max_n_alns should >= acutal_n_alns(%d)\n", gpu_storage->max_n_alns, actual_n_alns);

		int i = 2;
		while ( (gpu_storage->max_n_alns * i) < actual_n_alns) i++;

		fprintf(stderr, "[GASAL] Allocating GPU memory for %d number of alignments\n", gpu_storage->max_n_alns*i);
		gpu_storage->max_n_alns = gpu_storage->max_n_alns * i;

		if (gpu_storage->query_batch_offsets != NULL) CHECKCUDAERROR(hipFree(gpu_storage->query_batch_offsets));
		if (gpu_storage->target_batch_offsets != NULL) CHECKCUDAERROR(hipFree(gpu_storage->target_batch_offsets));
		if (gpu_storage->query_batch_lens != NULL) CHECKCUDAERROR(hipFree(gpu_storage->query_batch_lens));
		if (gpu_storage->target_batch_lens != NULL) CHECKCUDAERROR(hipFree(gpu_storage->target_batch_lens));
		if (gpu_storage->aln_score != NULL) CHECKCUDAERROR(hipFree(gpu_storage->aln_score));
		if (gpu_storage->query_batch_start != NULL) CHECKCUDAERROR(hipFree(gpu_storage->query_batch_start));
		if (gpu_storage->target_batch_start != NULL) CHECKCUDAERROR(hipFree(gpu_storage->target_batch_start));
		if (gpu_storage->query_batch_end != NULL) CHECKCUDAERROR(hipFree(gpu_storage->query_batch_end));
		if (gpu_storage->target_batch_end != NULL) CHECKCUDAERROR(hipFree(gpu_storage->target_batch_end));

		CHECKCUDAERROR(hipMalloc(&(gpu_storage->query_batch_lens), gpu_storage->max_n_alns * sizeof(uint32_t)));
		CHECKCUDAERROR(hipMalloc(&(gpu_storage->target_batch_lens), gpu_storage->max_n_alns * sizeof(uint32_t)));
		CHECKCUDAERROR(hipMalloc(&(gpu_storage->query_batch_offsets), gpu_storage->max_n_alns * sizeof(uint32_t)));
		CHECKCUDAERROR(hipMalloc(&(gpu_storage->target_batch_offsets), gpu_storage->max_n_alns * sizeof(uint32_t)));

		CHECKCUDAERROR(hipMalloc(&(gpu_storage->aln_score),gpu_storage->max_n_alns * sizeof(int32_t)));
		if (algo == GLOBAL) {
			gpu_storage->query_batch_start = NULL;
			gpu_storage->query_batch_end = NULL;
			gpu_storage->target_batch_start = NULL;
			gpu_storage->target_batch_end = NULL;
		} else {
			CHECKCUDAERROR(
					hipMalloc(&(gpu_storage->target_batch_end),
							gpu_storage->max_n_alns * sizeof(uint32_t)));
			if (start == WITH_START) {
				CHECKCUDAERROR(
						hipMalloc(&(gpu_storage->target_batch_start),
								gpu_storage->max_n_alns * sizeof(uint32_t)));
			} else
				gpu_storage->target_batch_start = NULL;
			if (algo == LOCAL) {
				CHECKCUDAERROR(
						hipMalloc(&(gpu_storage->query_batch_end),
								gpu_storage->max_n_alns * sizeof(uint32_t)));
				if (start == WITH_START) {
					CHECKCUDAERROR(
							hipMalloc(&(gpu_storage->query_batch_start),
									gpu_storage->max_n_alns * sizeof(uint32_t)));
				} else
					gpu_storage->query_batch_start = NULL;
			} else {
				gpu_storage->query_batch_start = NULL;
				gpu_storage->query_batch_end = NULL;
			}
		}



	}
	//-------------------------------------------------------------------------------------------

	//------------------------copy sequence batches from CPU to GPU---------------------------
	CHECKCUDAERROR(hipMemcpy(gpu_storage->unpacked_query_batch, query_batch, actual_query_batch_bytes, hipMemcpyHostToDevice));
	CHECKCUDAERROR(hipMemcpy(gpu_storage->unpacked_target_batch, target_batch, actual_target_batch_bytes, hipMemcpyHostToDevice));
	//----------------------------------------------------------------------------------------

    uint32_t BLOCKDIM = 128;
    uint32_t N_BLOCKS = (actual_n_alns + BLOCKDIM - 1) / BLOCKDIM;

    int query_batch_tasks_per_thread = (int)ceil((double)actual_query_batch_bytes/(8*BLOCKDIM*N_BLOCKS));
    int target_batch_tasks_per_thread = (int)ceil((double)actual_target_batch_bytes/(8*BLOCKDIM*N_BLOCKS));

    //launch packing kernel
    gasal_pack_kernel<<<N_BLOCKS, BLOCKDIM>>>((uint32_t*)(gpu_storage->unpacked_query_batch),
    						(uint32_t*)(gpu_storage->unpacked_target_batch), gpu_storage->packed_query_batch, gpu_storage->packed_target_batch,
    					    query_batch_tasks_per_thread, target_batch_tasks_per_thread, actual_query_batch_bytes/4, actual_target_batch_bytes/4);
    hipError_t pack_kernel_err = hipGetLastError();
    if ( hipSuccess != pack_kernel_err )
    {
    	 fprintf(stderr, "[GASAL] CUDA error:%d(%s) at line no. %d in file %s\n", pack_kernel_err, hipGetErrorString(pack_kernel_err), __LINE__, __FILE__);
         exit(EXIT_FAILURE);
    }

    //----------------------copy sequence offsetss and lengths from CPU to GPU--------------------------------------
    CHECKCUDAERROR(hipMemcpy(gpu_storage->query_batch_lens, query_batch_lens, actual_n_alns * sizeof(uint32_t), hipMemcpyHostToDevice));
    CHECKCUDAERROR(hipMemcpy(gpu_storage->target_batch_lens, target_batch_lens, actual_n_alns * sizeof(uint32_t), hipMemcpyHostToDevice));
    CHECKCUDAERROR(hipMemcpy(gpu_storage->query_batch_offsets, query_batch_offsets, actual_n_alns * sizeof(uint32_t), hipMemcpyHostToDevice));
    CHECKCUDAERROR(hipMemcpy(gpu_storage->target_batch_offsets, target_batch_offsetss, actual_n_alns * sizeof(uint32_t), hipMemcpyHostToDevice));
    //------------------------------------------------------------------------------------------------------------------------

    //--------------------------------------launch alignment kernels--------------------------------------------------------------
    if(algo == LOCAL) {
    	if (start == WITH_START) {
    		gasal_local_with_start_kernel<<<N_BLOCKS, BLOCKDIM>>>(gpu_storage->packed_query_batch, gpu_storage->packed_target_batch, gpu_storage->query_batch_lens,
    				gpu_storage->target_batch_lens, gpu_storage->query_batch_offsets, gpu_storage->target_batch_offsets, gpu_storage->aln_score,
    				gpu_storage->query_batch_end, gpu_storage->target_batch_end, gpu_storage->query_batch_start,
    				gpu_storage->target_batch_start, actual_n_alns);
    	} else {
    		gasal_local_kernel<<<N_BLOCKS, BLOCKDIM>>>(gpu_storage->packed_query_batch, gpu_storage->packed_target_batch, gpu_storage->query_batch_lens,
    				gpu_storage->target_batch_lens, gpu_storage->query_batch_offsets, gpu_storage->target_batch_offsets, gpu_storage->aln_score,
    				gpu_storage->query_batch_end, gpu_storage->target_batch_end, actual_n_alns);
    	}
    } else if (algo == SEMI_GLOBAL) {
    	if (start == WITH_START) {
    		gasal_semi_global_with_start_kernel<<<N_BLOCKS, BLOCKDIM>>>(gpu_storage->packed_query_batch, gpu_storage->packed_target_batch, gpu_storage->query_batch_lens,
    				gpu_storage->target_batch_lens, gpu_storage->query_batch_offsets, gpu_storage->target_batch_offsets, gpu_storage->aln_score, gpu_storage->target_batch_end,
    				gpu_storage->target_batch_start, actual_n_alns);
    	} else {
    		gasal_semi_global_kernel<<<N_BLOCKS, BLOCKDIM>>>(gpu_storage->packed_query_batch, gpu_storage->packed_target_batch, gpu_storage->query_batch_lens,
    				gpu_storage->target_batch_lens, gpu_storage->query_batch_offsets, gpu_storage->target_batch_offsets, gpu_storage->aln_score, gpu_storage->target_batch_end,
    				actual_n_alns);
    	}

    } else if (algo == GLOBAL) {
    	gasal_global_kernel<<<N_BLOCKS, BLOCKDIM>>>(gpu_storage->packed_query_batch, gpu_storage->packed_target_batch, gpu_storage->query_batch_lens,
    			gpu_storage->target_batch_lens, gpu_storage->query_batch_offsets, gpu_storage->target_batch_offsets, gpu_storage->aln_score, actual_n_alns);
    }
    else {
    	fprintf(stderr, "[GASAL] Algo type invalid\n");
    	exit(EXIT_FAILURE);
    }
    //-----------------------------------------------------------------------------------------------------------------------
    hipError_t aln_kernel_err = hipGetLastError();
    if ( hipSuccess != aln_kernel_err )
    {
    	fprintf(stderr, "[GASAL] CUDA error:%d(%s) at line no. %d in file %s\n", aln_kernel_err, hipGetErrorString(aln_kernel_err), __LINE__, __FILE__);
    	exit(EXIT_FAILURE);
    }

    //------------------------copy alignment results from GPU to CPU--------------------------------------
    if (host_aln_score != NULL && gpu_storage->aln_score != NULL) CHECKCUDAERROR(hipMemcpy(host_aln_score, gpu_storage->aln_score, actual_n_alns * sizeof(int32_t), hipMemcpyDeviceToHost));
    else {
    	fprintf(stderr, "[GASAL] The *host_aln_score input can't be NULL\n");
    	exit(EXIT_FAILURE);
    }
    if (host_query_batch_start != NULL && gpu_storage->query_batch_start != NULL) CHECKCUDAERROR(hipMemcpy(host_query_batch_start, gpu_storage->query_batch_start, actual_n_alns * sizeof(int32_t), hipMemcpyDeviceToHost));
    if (host_target_batch_start != NULL && gpu_storage->target_batch_start != NULL) CHECKCUDAERROR(hipMemcpy(host_target_batch_start, gpu_storage->target_batch_start, actual_n_alns * sizeof(int32_t), hipMemcpyDeviceToHost));
    if (host_query_batch_end != NULL && gpu_storage->query_batch_end != NULL) CHECKCUDAERROR(hipMemcpy(host_query_batch_end, gpu_storage->query_batch_end, actual_n_alns * sizeof(int32_t), hipMemcpyDeviceToHost));
    if (host_target_batch_end != NULL && gpu_storage->target_batch_end != NULL) CHECKCUDAERROR(hipMemcpy(host_target_batch_end, gpu_storage->target_batch_end, actual_n_alns * sizeof(int32_t), hipMemcpyDeviceToHost));
    //------------------------------------------------------------------------------------------------------

}






void gasal_gpu_mem_alloc(gasal_gpu_storage_t *gpu_storage, int max_query_batch_bytes, int max_target_batch_bytes, int max_n_alns, int algo, int start) {

	hipError_t err;
	if (gpu_storage->max_query_batch_bytes % 8) {
		fprintf(stderr, "max_query_batch_bytes should be multiple of 8\n");
		exit(EXIT_FAILURE);
	}
	if (gpu_storage->max_target_batch_bytes % 8) {
		fprintf(stderr, "max_target_batch_bytes should be multiple of 8\n");
		exit(EXIT_FAILURE);
	}
	CHECKCUDAERROR(hipMalloc(&(gpu_storage->unpacked_query_batch), max_query_batch_bytes * sizeof(uint8_t)));
	CHECKCUDAERROR(hipMalloc(&(gpu_storage->unpacked_target_batch), max_target_batch_bytes * sizeof(uint8_t)));

	CHECKCUDAERROR(hipMalloc(&(gpu_storage->packed_query_batch), (max_query_batch_bytes/8) * sizeof(uint32_t)));
	CHECKCUDAERROR(hipMalloc(&(gpu_storage->packed_target_batch), (max_target_batch_bytes/8) * sizeof(uint32_t)));

	CHECKCUDAERROR(hipMalloc(&(gpu_storage->query_batch_lens), max_n_alns * sizeof(uint32_t)));
	CHECKCUDAERROR(hipMalloc(&(gpu_storage->target_batch_lens), max_n_alns * sizeof(uint32_t)));
	CHECKCUDAERROR(hipMalloc(&(gpu_storage->query_batch_offsets), max_n_alns * sizeof(uint32_t)));
	CHECKCUDAERROR(hipMalloc(&(gpu_storage->target_batch_offsets), max_n_alns * sizeof(uint32_t)));

	CHECKCUDAERROR(hipMalloc(&(gpu_storage->aln_score), max_n_alns * sizeof(int32_t)));
	if (algo == GLOBAL) {
		gpu_storage->query_batch_start = NULL;
		gpu_storage->query_batch_end = NULL;
		gpu_storage->target_batch_start = NULL;
		gpu_storage->target_batch_end = NULL;
	} else {
		CHECKCUDAERROR(
				hipMalloc(&(gpu_storage->target_batch_end),
						max_n_alns * sizeof(uint32_t)));
		if (start == WITH_START) {
			CHECKCUDAERROR(
					hipMalloc(&(gpu_storage->target_batch_start),
							max_n_alns * sizeof(uint32_t)));
		} else
			gpu_storage->target_batch_start = NULL;
		if (algo == LOCAL) {
			CHECKCUDAERROR(
					hipMalloc(&(gpu_storage->query_batch_end),
							max_n_alns * sizeof(uint32_t)));
			if (start == WITH_START) {
				CHECKCUDAERROR(
						hipMalloc(&(gpu_storage->query_batch_start),
								max_n_alns * sizeof(uint32_t)));
			} else
				gpu_storage->query_batch_start = NULL;
		} else {
			gpu_storage->query_batch_start = NULL;
			gpu_storage->query_batch_end = NULL;
		}
	}

	gpu_storage->max_query_batch_bytes = max_query_batch_bytes;
	gpu_storage->max_target_batch_bytes = max_target_batch_bytes;
	gpu_storage->max_n_alns = max_n_alns;

}




void gasal_gpu_mem_free(gasal_gpu_storage_t *gpu_storage) {

	hipError_t err;

	if (gpu_storage->unpacked_query_batch != NULL) CHECKCUDAERROR(hipFree(gpu_storage->unpacked_query_batch));
	if (gpu_storage->unpacked_target_batch != NULL) CHECKCUDAERROR(hipFree(gpu_storage->unpacked_target_batch));
	if (gpu_storage->packed_query_batch != NULL) CHECKCUDAERROR(hipFree(gpu_storage->packed_query_batch));
	if (gpu_storage->packed_target_batch != NULL) CHECKCUDAERROR(hipFree(gpu_storage->packed_target_batch));
	if (gpu_storage->query_batch_offsets != NULL) CHECKCUDAERROR(hipFree(gpu_storage->query_batch_offsets));
	if (gpu_storage->target_batch_offsets != NULL) CHECKCUDAERROR(hipFree(gpu_storage->target_batch_offsets));
	if (gpu_storage->query_batch_lens != NULL) CHECKCUDAERROR(hipFree(gpu_storage->query_batch_lens));
	if (gpu_storage->target_batch_lens != NULL) CHECKCUDAERROR(hipFree(gpu_storage->target_batch_lens));
	if (gpu_storage->aln_score != NULL) CHECKCUDAERROR(hipFree(gpu_storage->aln_score));
	if (gpu_storage->query_batch_start != NULL) CHECKCUDAERROR(hipFree(gpu_storage->query_batch_start));
	if (gpu_storage->target_batch_start != NULL) CHECKCUDAERROR(hipFree(gpu_storage->target_batch_start));
	if (gpu_storage->query_batch_end != NULL) CHECKCUDAERROR(hipFree(gpu_storage->query_batch_end));
	if (gpu_storage->target_batch_end != NULL) CHECKCUDAERROR(hipFree(gpu_storage->target_batch_end));

}


void gasal_copy_subst_scores(gasal_subst_scores *subst){

	hipError_t err;
	CHECKCUDAERROR(hipMemcpyToSymbol(HIP_SYMBOL(_cudaGapO), &(subst->gap_open), sizeof(int32_t), 0, hipMemcpyHostToDevice));
	CHECKCUDAERROR(hipMemcpyToSymbol(HIP_SYMBOL(_cudaGapExtend), &(subst->gap_extend), sizeof(int32_t), 0, hipMemcpyHostToDevice));
	int32_t gapoe = subst->gap_open + subst->gap_extend;
	CHECKCUDAERROR(hipMemcpyToSymbol(HIP_SYMBOL(_cudaGapOE), &(gapoe), sizeof(int32_t), 0, hipMemcpyHostToDevice));
	CHECKCUDAERROR(hipMemcpyToSymbol(HIP_SYMBOL(_cudaMatchScore), &(subst->match), sizeof(int32_t), 0, hipMemcpyHostToDevice));
	CHECKCUDAERROR(hipMemcpyToSymbol(HIP_SYMBOL(_cudaMismatchScore), &(subst->mismatch), sizeof(int32_t), 0, hipMemcpyHostToDevice));
	return;
}




